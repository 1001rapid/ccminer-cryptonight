#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cryptonight.h"

#ifndef _WIN32
#include <unistd.h>
#endif

extern int device_arch[8][2];
extern int device_bfactor[8];
extern int device_bsleep[8];

#include "cuda_cryptonight_aes.cu"

__device__ __forceinline__ uint64_t cuda_mul128(uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi)
{
	*product_hi = __umul64hi(multiplier, multiplicand);
	return(multiplier * multiplicand);
}

__global__ void cryptonight_core_gpu_phase1(int threads, uint32_t * __restrict__ long_state, uint32_t * __restrict__ ctx_state, uint32_t * __restrict__ ctx_key1)
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);

	const int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
	const int sub = (threadIdx.x & 7) << 2;

	if(thread < threads)
	{
		uint32_t key[40], text[4];

		MEMCPY8(key, ctx_key1 + thread * 40, 20);
		MEMCPY8(text, ctx_state + thread * 50 + sub + 16, 2);

		__syncthreads();
		for(int i = 0; i < 0x80000; i += 32)
		{
			cn_aes_pseudo_round_mut(sharedMemory, text, key);
			MEMCPY8(&long_state[(thread << 19) + sub + i], text, 2);
		}
	}
}

__device__ __forceinline__ void MUL_SUM_XOR_DST(const uint64_t *__restrict__ a, uint64_t *__restrict__ c, uint64_t *__restrict__ dst)
{
	uint64_t hi, lo = cuda_mul128(a[0], dst[0], &hi) + c[1];
	hi += c[0];
	c[0] = dst[0] ^ hi;
	c[1] = dst[1] ^ lo;
	dst[0] = hi;
	dst[1] = lo;
}

__global__ void cryptonight_core_gpu_phase2(uint32_t threads, int bfactor, int partidx, uint32_t * __restrict__ d_long_state, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b)
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);

	__syncthreads();

#if __CUDA_ARCH__ >= 300 && __CUDA_ARCH__ < 600

	const int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
	const int sub = threadIdx.x & 3;

	if(thread < threads)
	{
		int i, j, k;
		const int batchsize = ITER >> (2 + bfactor);
		const int start = partidx * batchsize;
		const int end = start + batchsize;
		uint32_t * __restrict__ long_state = &d_long_state[thread << 19];
		uint32_t * __restrict__ ctx_a = d_ctx_a + thread * 4;
		uint32_t * __restrict__ ctx_b = d_ctx_b + thread * 4;
		uint32_t a, b, c, x[4];
		uint32_t t1[4], t2[4], res;
		uint64_t reshi, reslo;

		a = ctx_a[sub];
		b = ctx_b[sub];

#pragma unroll 8
		for(i = start; i < end; ++i)
		{

			//j = ((uint32_t *)a)[0] & 0x1FFFF0;
			j = (__shfl((int)a, 0, 4) & 0x1FFFF0) >> 2;

			//cn_aes_single_round(sharedMemory, &long_state[j], c, a);
			x[0] = long_state[j + sub];
			x[1] = __shfl((int)x[0], sub + 1, 4);
			x[2] = __shfl((int)x[0], sub + 2, 4);
			x[3] = __shfl((int)x[0], sub + 3, 4);
			c = a ^
				t_fn0(x[0] & 0xff) ^
				t_fn1((x[1] >> 8) & 0xff) ^
				t_fn2((x[2] >> 16) & 0xff) ^
				t_fn3((x[3] >> 24) & 0xff);

			//XOR_BLOCKS_DST(c, b, &long_state[j]);
			long_state[j + sub] = c ^ b;

			//MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & 0x1FFFF0]);
			j = (__shfl((int)c, 0, 4) & 0x1FFFF0) >> 2;
#pragma unroll
			for(k = 0; k < 2; k++)
				t1[k] = __shfl((int)c, k, 4);
#pragma unroll
			for(k = 0; k < 4; k++)
				t2[k] = __shfl((int)a, k, 4);
			asm(
				"mad.lo.u64 %0, %2, %3, %4;\n\t"
				"mad.hi.u64 %1, %2, %3, %5;\n\t"
				: "=l"(reslo), "=l"(reshi)
				: "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)long_state)[j >> 1]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
			res = (sub & 2 ? reslo : reshi) >> (sub & 1 ? 32 : 0);
			a = long_state[j + sub] ^ res;
			long_state[j + sub] = res;

			//j = ((uint32_t *)a)[0] & 0x1FFFF0;
			j = (__shfl((int)a, 0, 4) & 0x1FFFF0) >> 2;

			//cn_aes_single_round(sharedMemory, &long_state[j], b, a);
			x[0] = long_state[j + sub];
			x[1] = __shfl((int)x[0], sub + 1, 4);
			x[2] = __shfl((int)x[0], sub + 2, 4);
			x[3] = __shfl((int)x[0], sub + 3, 4);
			b = a ^
				t_fn0(x[0] & 0xff) ^
				t_fn1((x[1] >> 8) & 0xff) ^
				t_fn2((x[2] >> 16) & 0xff) ^
				t_fn3((x[3] >> 24) & 0xff);

			//XOR_BLOCKS_DST(b, c, &long_state[j]);
			long_state[j + sub] = c ^ b;

			//MUL_SUM_XOR_DST(b, a, &long_state[((uint32_t *)b)[0] & 0x1FFFF0]);
			j = (__shfl((int)b, 0, 4) & 0x1FFFF0) >> 2;
#pragma unroll
			for(k = 0; k < 2; k++)
				t1[k] = __shfl((int)b, k, 4);
#pragma unroll
			for(k = 0; k < 4; k++)
				t2[k] = __shfl((int)a, k, 4);
			asm(
				"mad.lo.u64 %0, %2, %3, %4;\n\t"
				"mad.hi.u64 %1, %2, %3, %5;\n\t"
				: "=l"(reslo), "=l"(reshi)
				: "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)long_state)[j >> 1]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
			res = (sub & 2 ? reslo : reshi) >> (sub & 1 ? 32 : 0);
			a = long_state[j + sub] ^ res;
			long_state[j + sub] = res;
		}

		if(bfactor > 0)
		{
			ctx_a[sub] = a;
			ctx_b[sub] = b;
		}
	}

#else // __CUDA_ARCH__ < 300
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	if(thread < threads)
	{
		int j;
		const int batchsize = ITER >> (2 + bfactor);
		const int start = partidx * batchsize;
		const int end = start + batchsize;
		uint32_t * __restrict__ long_state = &d_long_state[thread << 19];
		uint64_t * __restrict__ ctx_a = (uint64_t*)(d_ctx_a + thread * 4);
		uint64_t * __restrict__ ctx_b = (uint64_t*)(d_ctx_b + thread * 4);
		uint64_t a[2], b[2], c[2];
		uint32_t *a32 = (uint32_t*)a;
		uint32_t *b32 = (uint32_t*)b;
		uint32_t *c32 = (uint32_t*)c;

		a[0] = ctx_a[0];
		a[1] = ctx_a[1];
		b[0] = ctx_b[0];
		b[1] = ctx_b[1];

		for(int i = start; i < end; ++i)
		{
			j = (a32[0] & 0x001FFFF0) >> 2;
			cn_aes_single_round(sharedMemory, &long_state[j], c32, a32);
			XOR_BLOCKS_DST2(c, b, &long_state[j]);
			MUL_SUM_XOR_DST(c, a, (uint64_t*)&long_state[(c[0] & 0x001FFFF0) >> 2]);
			j = (((uint32_t*)a)[0] & 0x1FFFF0) >> 2;
			cn_aes_single_round(sharedMemory, &long_state[j], b32, a32);
			XOR_BLOCKS_DST2(b, c, &long_state[j]);
			MUL_SUM_XOR_DST(b, a, (uint64_t*)&long_state[(b[0] & 0x1FFFF0) >> 2]);
		}

		if(bfactor > 0)
		{

			ctx_a[0] = a[0];
			ctx_a[1] = a[1];
			ctx_b[0] = b[0];
			ctx_b[1] = b[1];
		}
	}
#endif // __CUDA_ARCH__ >= 300
}

__global__ void cryptonight_core_gpu_phase3(int threads, const uint32_t * __restrict__ long_state, uint32_t * __restrict__ d_ctx_state, uint32_t * __restrict__ d_ctx_key2)
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);

	int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
	int sub = (threadIdx.x & 7) << 2;

	if(thread < threads)
	{
		uint32_t key[40], text[4], i, j;
		MEMCPY8(key, d_ctx_key2 + thread * 40, 20);
		MEMCPY8(text, d_ctx_state + thread * 50 + sub + 16, 2);

		__syncthreads();
		for(i = 0; i < 0x80000; i += 32)
		{
#pragma unroll
			for(j = 0; j < 4; ++j)
				text[j] ^= long_state[(thread << 19) + sub + i + j];

			cn_aes_pseudo_round_mut(sharedMemory, text, key);
		}

		MEMCPY8(d_ctx_state + thread * 50 + sub + 16, text, 2);
	}
}

__host__ void cryptonight_core_cpu_hash(int thr_id, int blocks, int threads, uint32_t *d_long_state, uint32_t *d_ctx_state, uint32_t *d_ctx_a, uint32_t *d_ctx_b, uint32_t *d_ctx_key1, uint32_t *d_ctx_key2)
{
	dim3 grid(blocks);
	dim3 block(threads);
	dim3 block4(threads << 2);
	dim3 block8(threads << 3);

	int i, partcount = 1 << device_bfactor[thr_id];

	cryptonight_core_gpu_phase1 << <grid, block8 >> >(blocks*threads, d_long_state, d_ctx_state, d_ctx_key1);
	exit_if_cudaerror(thr_id, __FILE__, __LINE__);
	if(partcount > 1) usleep(device_bsleep[thr_id]);

	for(i = 0; i < partcount; i++)
	{
		cryptonight_core_gpu_phase2 << <grid, ((device_arch[thr_id][0] == 3 || device_arch[thr_id][0] == 5) ? block4 : block)>> >(blocks*threads, device_bfactor[thr_id], i, d_long_state, d_ctx_a, d_ctx_b);
		exit_if_cudaerror(thr_id, __FILE__, __LINE__);
		if(partcount > 1) usleep(device_bsleep[thr_id]);
	}

	cryptonight_core_gpu_phase3 << <grid, block8 >> >(blocks*threads, d_long_state, d_ctx_state, d_ctx_key2);
	exit_if_cudaerror(thr_id, __FILE__, __LINE__);
}
