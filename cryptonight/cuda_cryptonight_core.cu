#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cryptonight.h"

#ifndef _WIN32
#include <unistd.h>
#endif

extern int device_arch[8][2];
extern int device_bfactor[8];
extern int device_bsleep[8];

#include "cuda_cryptonight_aes.cu"

__device__ __forceinline__ uint64_t cuda_mul128(uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi)
{
    *product_hi = __umul64hi(multiplier, multiplicand);
    return(multiplier * multiplicand);
}

__global__ void cryptonight_core_gpu_phase1(int threads, uint8_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
	__shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

	__syncthreads();

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
    int sub = threadIdx.x & 7;
   
    if (thread < threads)
    {
        int i, j;
        int start = 0, end = MEMORY;
        uint8_t *long_state = &d_long_state[MEMORY * thread];
        uint32_t *ls32;
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t key[40];
        uint32_t text[4];
        uint32_t *state = (uint32_t *)&ctx->state[16+(sub<<2)];

        MEMCPY8(key, ctx->key1, 20);
        for( i = 0; i < 4; i++ )
            text[i] = state[i];

        for (i = start; i < end; i += INIT_SIZE_BYTE) {

            ls32 = (uint32_t *)&long_state[i];

            cn_aes_pseudo_round_mut(sharedMemory, text, key);

            for( j = 0; j < 4; j++ )
                ls32[(sub<<2) + j] = text[j];
        }
    }
}

__global__ void cryptonight_core_gpu_phase2(int threads, int bfactor, int partidx, uint8_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
	__shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

	__syncthreads();

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;

#if __CUDA_ARCH__ >= 300

    int sub = threadIdx.x & 3;

    if (thread < threads)
    {
        int i, j, k;
        int batchsize = ITER >> (2+bfactor);
        int start = partidx * batchsize;
        int end = start + batchsize;
        uint8_t *long_state = &d_long_state[MEMORY * thread];
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t a, b, c, x[4];
        uint32_t *ls32;
        uint32_t t1[4], t2[4], res;
        uint64_t reshi, reslo;

        a = ctx->a[sub];
        b = ctx->b[sub];

        for (i = start; i < end; ++i) {

            //j = ((uint32_t *)a)[0] & 0x1FFFF0;
            j = __shfl((int)a, 0, 4) & 0x1FFFF0;
            
            //cn_aes_single_round(sharedMemory, &long_state[j], c, a);
            ls32 = (uint32_t *)&long_state[j];
            x[0] = ls32[sub];
            x[1] = __shfl((int)x[0], sub+1, 4);
            x[2] = __shfl((int)x[0], sub+2, 4);
            x[3] = __shfl((int)x[0], sub+3, 4);
            c = a ^
                t_fn0(x[0] & 0xff) ^
                t_fn1((x[1] >> 8) & 0xff) ^
                t_fn2((x[2] >> 16) & 0xff) ^
                t_fn3((x[3] >> 24) & 0xff);
            
            //XOR_BLOCKS_DST(c, b, &long_state[j]);
            ls32[sub] = c ^ b;

            //MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & 0x1FFFF0]);
            ls32 = (uint32_t *)&long_state[__shfl((int)c, 0, 4) & 0x1FFFF0];
            for( k = 0; k < 2; k++ ) t1[k] = __shfl((int)c, k, 4);
            for( k = 0; k < 4; k++ ) t2[k] = __shfl((int)a, k, 4);
            asm(
                "mad.lo.u64 %0, %2, %3, %4;\n\t"
                "mad.hi.u64 %1, %2, %3, %5;\n\t"
                : "=l"(reslo), "=l"(reshi)
                : "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)ls32)[0]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
            res = (sub & 2 ? reslo : reshi) >> (sub&1 ? 32 : 0);
            a = ls32[sub] ^ res;
            ls32[sub] = res;

            //j = ((uint32_t *)a)[0] & 0x1FFFF0;
            j = __shfl((int)a, 0, 4) & 0x1FFFF0;
            
            //cn_aes_single_round(sharedMemory, &long_state[j], b, a);
            ls32 = (uint32_t *)&long_state[j];
            x[0] = ls32[sub];
            x[1] = __shfl((int)x[0], sub+1, 4);
            x[2] = __shfl((int)x[0], sub+2, 4);
            x[3] = __shfl((int)x[0], sub+3, 4);
            b = a ^
                t_fn0(x[0] & 0xff) ^
                t_fn1((x[1] >> 8) & 0xff) ^
                t_fn2((x[2] >> 16) & 0xff) ^
                t_fn3((x[3] >> 24) & 0xff);

            //XOR_BLOCKS_DST(b, c, &long_state[j]);
            ls32[sub] = c ^ b;

            //MUL_SUM_XOR_DST(b, a, &long_state[((uint32_t *)b)[0] & 0x1FFFF0]);
            ls32 = (uint32_t *)&long_state[__shfl((int)b, 0, 4) & 0x1FFFF0];
            for( k = 0; k < 2; k++ ) t1[k] = __shfl((int)b, k, 4);
            for( k = 0; k < 4; k++ ) t2[k] = __shfl((int)a, k, 4);
            asm(
                "mad.lo.u64 %0, %2, %3, %4;\n\t"
                "mad.hi.u64 %1, %2, %3, %5;\n\t"
                : "=l"(reslo), "=l"(reshi)
                : "l"(((uint64_t *)t1)[0]), "l"(((uint64_t *)ls32)[0]), "l"(((uint64_t *)t2)[1]), "l"(((uint64_t *)t2)[0]));
            res = (sub & 2 ? reslo : reshi) >> (sub&1 ? 32 : 0);
            a = ls32[sub] ^ res;
            ls32[sub] = res;
        }

        if( bfactor > 0 ) {

            ctx->a[sub] = a;
            ctx->b[sub] = b;
        }
    }

#else // __CUDA_ARCH__ < 300

    if (thread < threads)
    {
        int i, j;
        int batchsize = ITER >> (2+bfactor);
        int start = partidx * batchsize;
        int end = start + batchsize;
        uint8_t *long_state = &d_long_state[MEMORY * thread];
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t a[4], b[4], c[4];

        MEMCPY8(a, ctx->a, 2);
        MEMCPY8(b, ctx->b, 2);

        for (i = start; i < end; ++i) {

            j = ((uint32_t *)a)[0] & 0x1FFFF0;
            cn_aes_single_round(sharedMemory, &long_state[j], c, a);
            XOR_BLOCKS_DST(c, b, &long_state[j]);
            MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & 0x1FFFF0]);
            j = ((uint32_t *)a)[0] & 0x1FFFF0;
            cn_aes_single_round(sharedMemory, &long_state[j], b, a);
            XOR_BLOCKS_DST(b, c, &long_state[j]);
            MUL_SUM_XOR_DST(b, a, &long_state[((uint32_t *)b)[0] & 0x1FFFF0]);
        }
        
        if( bfactor > 0 ) {

            MEMCPY8(ctx->a, a, 2);
            MEMCPY8(ctx->b, b, 2);
        }
    }

#endif // __CUDA_ARCH__ >= 300
}

__global__ void cryptonight_core_gpu_phase3(int threads, uint8_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
	__shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

	__syncthreads();

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
    int sub = threadIdx.x & 7;
   
    if (thread < threads)
    {
        int i, j;
        int start = 0, end = MEMORY;
        uint8_t *long_state = &d_long_state[MEMORY * thread];
        uint32_t *ls32;
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t key[40];
        uint32_t text[4];
        uint32_t *state = (uint32_t *)&ctx->state[16+(sub<<2)];

        MEMCPY8(key, ctx->key2, 20);
        for( i = 0; i < 4; i++ )
            text[i] = state[i];

        for (i = start; i < end; i += INIT_SIZE_BYTE) {

            ls32 = (uint32_t *)&long_state[i];

            for( j = 0; j < 4; j++ )
                text[j] ^= ls32[(sub<<2)+j];

            cn_aes_pseudo_round_mut(sharedMemory, text, key);
        }

        for( i = 0; i < 4; i++ )
            state[i] = text[i];
    }
}


__host__ void cryptonight_core_cpu_init(int thr_id, int threads)
{
	cn_aes_cpu_init();
}

__host__ void cryptonight_core_cpu_hash(int thr_id, int blocks, int threads, uint8_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
    dim3 grid(blocks);
    dim3 block(threads);
    dim3 block4(threads << 2);
    dim3 block8(threads << 3);

    size_t shared_size = 1024;
    int i, partcount = 1 << device_bfactor[thr_id];

    cryptonight_core_gpu_phase1<<<grid, block8, shared_size>>>(blocks*threads, d_long_state, d_ctx);
    hipDeviceSynchronize();
    if( partcount > 1 ) usleep(device_bsleep[thr_id]);

    for( i = 0; i < partcount; i++ ) {
        cryptonight_core_gpu_phase2<<<grid, (device_arch[thr_id][0] >= 3 ? block4 : block), shared_size>>>(blocks*threads, device_bfactor[thr_id], i, d_long_state, d_ctx);
        hipDeviceSynchronize();
        if( partcount > 1 ) usleep(device_bsleep[thr_id]);
    }

    cryptonight_core_gpu_phase3<<<grid, block8, shared_size>>>(blocks*threads, d_long_state, d_ctx);
    hipDeviceSynchronize();
}
