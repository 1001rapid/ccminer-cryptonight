#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cryptonight.h"

#ifndef _WIN32
#include <unistd.h>
#endif

#include "cuda_cryptonight_aes.cu"

#define hi_dword(x) (x >> 32)
#define lo_dword(x) (x & 0xFFFFFFFF)

__device__ __forceinline__ uint64_t cuda_mul128(uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi)
{
  uint64_t a = hi_dword(multiplier);
  uint64_t b = lo_dword(multiplier);
  uint64_t c = hi_dword(multiplicand);
  uint64_t d = lo_dword(multiplicand);

  uint64_t ac = a * c;
  uint64_t ad = a * d;
  uint64_t bc = b * c;
  uint64_t bd = b * d;

  uint64_t adbc = ad + bc;
  uint64_t adbc_carry = adbc < ad ? 1 : 0;

  uint64_t product_lo = bd + (adbc << 32);
  uint64_t product_lo_carry = product_lo < bd ? 1 : 0;
  *product_hi = ac + (adbc >> 32) + (adbc_carry << 32) + product_lo_carry;

  return product_lo;
}

__global__ void cryptonight_core_gpu_phase1(int threads, uint8_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
	__shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

	__syncthreads();

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
    int sub = threadIdx.x & 7;
   
    if (thread < threads)
    {
        int i, j;
        uint8_t *long_state = &d_long_state[MEMORY * thread];
        uint32_t *ls32;
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t key[40];
        uint32_t text[4];
        uint32_t *state = (uint32_t *)&ctx->state[16+(sub<<2)];

        MEMCPY8(key, ctx->key1, 20);
        for( i = 0; i < 4; i++ )
            text[i] = state[i];

        for (i = 0; i < MEMORY; i += INIT_SIZE_BYTE) {

            ls32 = (uint32_t *)&long_state[i];

            cn_aes_pseudo_round_mut(sharedMemory, text, key);

            for( j = 0; j < 4; j++ )
                ls32[(sub<<2) + j] = text[j];
        }
    }
}

__global__ void cryptonight_core_gpu_phase2(int threads, uint8_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
	__shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

	__syncthreads();

    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
   
    if (thread < threads)
    {
        int i, j;
        uint8_t *long_state = &d_long_state[MEMORY * thread];
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t a[4], b[4], c[4];

        MEMCPY8(a, ctx->a, 2);
        MEMCPY8(b, ctx->b, 2);

        for (i = 0; i < ITER / 4; ++i) {

            j = E2I(a) * AES_BLOCK_SIZE;
            cn_aes_single_round(sharedMemory, &long_state[j], c, a);
            XOR_BLOCKS_DST(c, b, &long_state[j]);
            MUL_SUM_XOR_DST(c, a, &long_state[E2I(c) * AES_BLOCK_SIZE]);
            j = E2I(a) * AES_BLOCK_SIZE;
            cn_aes_single_round(sharedMemory, &long_state[j], b, a);
            XOR_BLOCKS_DST(b, c, &long_state[j]);
            MUL_SUM_XOR_DST(b, a, &long_state[E2I(b) * AES_BLOCK_SIZE]);
        }
    }
}

__global__ void cryptonight_core_gpu_phase3(int threads, uint8_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
	__shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init(sharedMemory);

	__syncthreads();

    int thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
    int sub = threadIdx.x & 7;
   
    if (thread < threads)
    {
        int i, j;
        uint8_t *long_state = &d_long_state[MEMORY * thread];
        uint32_t *ls32;
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t key[40];
        uint32_t text[4];
        uint32_t *state = (uint32_t *)&ctx->state[16+(sub<<2)];

        MEMCPY8(key, ctx->key2, 20);
        for( i = 0; i < 4; i++ )
            text[i] = state[i];

        for (i = 0; i < MEMORY; i += INIT_SIZE_BYTE) {

            ls32 = (uint32_t *)&long_state[i];

            for( j = 0; j < 4; j++ )
                text[j] ^= ls32[(sub<<2)+j];

            cn_aes_pseudo_round_mut(sharedMemory, text, key);
        }

        for( i = 0; i < 4; i++ )
            state[i] = text[i];
    }
}

__host__ void cryptonight_core_cpu_init(int thr_id, int threads)
{
	cn_aes_cpu_init();
}

__host__ void cryptonight_core_cpu_hash(int thr_id, int blocks, int threads, uint8_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx)
{
    dim3 grid(blocks);
    dim3 block(threads);
    dim3 block8(threads << 3);

    size_t shared_size = 1024;

    cryptonight_core_gpu_phase1<<<grid, block8, shared_size>>>(blocks*threads, d_long_state, d_ctx);
    hipDeviceSynchronize();

    cryptonight_core_gpu_phase2<<<grid, block, shared_size>>>(blocks*threads, d_long_state, d_ctx);
    hipDeviceSynchronize();

    cryptonight_core_gpu_phase3<<<grid, block8, shared_size>>>(blocks*threads, d_long_state, d_ctx);
    hipDeviceSynchronize();
}
