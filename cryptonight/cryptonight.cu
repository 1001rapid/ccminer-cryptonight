#include <unistd.h>
#include <stdio.h>
#include <stdint.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

extern "C"
{
#include "cpuminer-config.h"
#include "miner.h"
#include "cryptonight.h"
}

extern int device_map[8];
extern int device_config[8][2];

static uint8_t *d_long_state[8];
static struct cryptonight_gpu_ctx *d_ctx[8];

extern bool opt_benchmark;

extern void cryptonight_core_cpu_init(int thr_id, int threads);
extern void cryptonight_core_cpu_hash(int thr_id, int blocks, int threads, uint8_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx);

extern void cryptonight_extra_cpu_setData(int thr_id, const void *data, const void *pTargetIn);
extern void cryptonight_extra_cpu_init(int thr_id);
extern void cryptonight_extra_cpu_prepare(int thr_id, int threads, uint32_t startNonce, struct cryptonight_gpu_ctx *d_ctx);
extern void cryptonight_extra_cpu_final(int thr_id, int threads, uint32_t startNonce, uint32_t *nonce, struct cryptonight_gpu_ctx *d_ctx);

extern "C" void cryptonight_hash(void* output, const void* input, size_t len);

extern "C" int scanhash_cryptonight(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
    uint32_t *nonceptr = (uint32_t*)(((char*)pdata) + 39);
    const uint32_t first_nonce = *nonceptr;
    uint32_t nonce = *nonceptr;
    int cn_blocks = device_config[thr_id][0], cn_threads = device_config[thr_id][1];

	if (opt_benchmark) {
		((uint32_t*)ptarget)[7] = 0x0000ff;
        pdata[17] = 0;
    }
	const uint32_t Htarg = ptarget[7];
	const int throughput = cn_threads * cn_blocks;
    const size_t alloc = MEMORY * throughput;

    static bool init[8] = { false, false, false, false, false, false, false, false };
	if (!init[thr_id])
	{
        hipSetDevice(device_map[thr_id]);
        hipDeviceReset();
        hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if( hipMalloc(&d_long_state[thr_id], alloc) != hipSuccess ) {
            applog(LOG_ERR, "GPU #%d: FATAL: failed to allocate device memory for long state", thr_id);
            exit(1);
        }
		if( hipMalloc(&d_ctx[thr_id], sizeof(struct cryptonight_gpu_ctx) * throughput) != hipSuccess ) {
            applog(LOG_ERR, "GPU #%d: FATAL: failed to allocate device memory for hash context", thr_id);
            exit(1);
        }
		cryptonight_core_cpu_init(thr_id, throughput);
        cryptonight_extra_cpu_init(thr_id);
		init[thr_id] = true;
	}

    cryptonight_extra_cpu_setData(thr_id, (const void *)pdata, (const void *)ptarget);

	do {
        uint32_t foundNonce = 0xFFFFFFFF;

        cryptonight_extra_cpu_prepare(thr_id, throughput, nonce, d_ctx[thr_id]);
        cryptonight_core_cpu_hash(thr_id, cn_blocks, cn_threads, d_long_state[thr_id], d_ctx[thr_id]);
        cryptonight_extra_cpu_final(thr_id, throughput, nonce, &foundNonce, d_ctx[thr_id]);

        if (foundNonce < 0xffffffff)
		{
			uint32_t vhash64[8];
            uint32_t tempdata[19];
            memcpy(tempdata, pdata, 76);
            uint32_t *tempnonceptr = (uint32_t*)(((char*)tempdata) + 39);
			*tempnonceptr = foundNonce;
			cryptonight_hash(vhash64, tempdata, 76);

            if( (vhash64[7] <= Htarg) && fulltest(vhash64, ptarget) ) {
                
			    *nonceptr = foundNonce;
                *hashes_done = foundNonce - first_nonce + 1;
                return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}
		
            foundNonce = 0xffffffff;
        }

		nonce += throughput;
	} while (nonce < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = nonce - first_nonce + 1;
	return 0;
}
