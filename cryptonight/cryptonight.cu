#include <unistd.h>
#include <stdio.h>
#include <stdint.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

extern "C"
{
#include "cpuminer-config.h"
#include "miner.h"
#include "cryptonight.h"
}

extern int device_map[8];
extern int device_config[8][2];

static uint8_t *d_long_state[8];
static union cn_gpu_hash_state *d_hash_state[8];

extern bool opt_benchmark;

extern void cryptonight_cpu_init(int thr_id, int threads);
extern void cryptonight_cpu_setInput(int thr_id, void *data, void *pTargetIn);
extern void cryptonight_cpu_hash(int thr_id, int blocks, int threads, uint32_t startNonce, uint32_t *nonce, uint8_t *d_long_state, union cn_gpu_hash_state *d_hash_state);

extern "C" void cryptonight_hash(void* output, const void* input, size_t len);

extern "C" int scanhash_cryptonight(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
    uint32_t *nonceptr = (uint32_t*)(((char*)pdata) + 39);
    const uint32_t first_nonce = *nonceptr;
    uint32_t nonce = *nonceptr;
    int cn_blocks = device_config[thr_id][0], cn_threads = device_config[thr_id][1];

	if (opt_benchmark) {
		((uint32_t*)ptarget)[7] = 0x0000ff;
        pdata[17] = 0;
    }
	const uint32_t Htarg = ptarget[7];
	const int throughput = cn_threads * cn_blocks;
    const size_t alloc = MEMORY * throughput;

    static bool init[8] = { false, false, false, false, false, false, false, false };
	if (!init[thr_id])
	{
        hipSetDevice(device_map[thr_id]);
        hipDeviceReset();
        hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if( hipMalloc(&d_long_state[thr_id], alloc) != hipSuccess ) {
            applog(LOG_ERR, "GPU #%d: FATAL: failed to allocate device memory for the long state variable", thr_id);
            exit(1);
        }
		if( hipMalloc(&d_hash_state[thr_id], sizeof(union cn_gpu_hash_state) * throughput) != hipSuccess ) {
            applog(LOG_ERR, "GPU #%d: FATAL: failed to allocate device memory for the hash state variable", thr_id);
            exit(1);
        }
		cryptonight_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

    cryptonight_cpu_setInput(thr_id, (void *)pdata, (void*)ptarget);

	do {
        uint32_t foundNonce = 0xFFFFFFFF;

        cryptonight_cpu_hash(thr_id, cn_blocks, cn_threads, nonce, &foundNonce, d_long_state[thr_id], d_hash_state[thr_id]);

        if (foundNonce < 0xffffffff)
		{
			uint32_t vhash64[8];
            uint32_t tempdata[19];
            memcpy(tempdata, pdata, 76);
            uint32_t *tempnonceptr = (uint32_t*)(((char*)tempdata) + 39);
			*tempnonceptr = foundNonce;
			cryptonight_hash(vhash64, tempdata, 76);

            if( (vhash64[7] <= Htarg) && fulltest(vhash64, ptarget) ) {
                
			    *nonceptr = foundNonce;
                *hashes_done = foundNonce - first_nonce + 1;
                return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}
		
            foundNonce = 0xffffffff;
        }

		nonce += throughput;
	} while (nonce < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = nonce - first_nonce + 1;
	return 0;
}
